#include "hip/hip_runtime.h"
#include "../src/kernels.h"
#include "../src/lanczos.h"
#include "../src/matrix-util.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>

#define BLOCK_SIZE 32
#define TRAILS 100
#define WARMUP 100

#define tcalloc(T, n) (T *)calloc(n, sizeof(T))
#define tfree(p) free_((void **)p)
void free_(void **p) { free(*p), *p = NULL; }

double *create_host_vec(int size) {
  double *x = tcalloc(double, size);
  for (int i = 0; i < size; i++)
    x[i] = (rand() + 1.0) / RAND_MAX;

  return x;
}

int inc(int i) {
  return (int)(1.01 * i);
  if (i < 1000)
    return i + 1;
  else
    return (int)(1.03 * i);
}

FILE *open_file(char *suffix) {
  char fname[2 * BUFSIZ];
  strncpy(fname, "lanczos", BUFSIZ);
  strncat(fname, "_", 2);
  strncat(fname, suffix, BUFSIZ);
  strncat(fname, ".txt", 5);

  FILE *fp = fopen(fname, "a");
  if (!fp)
    printf("Not found \n");
  return fp;
}
void vec_sclr_mul_bench() {
  FILE *fp = open_file("vec-sclr-mul-cuda");
  for (int i = 1e4; i < 1e7; i = inc(i)) {
    double *h_a = create_host_vec(i);

    double *d_a, *d_b;

    // Allocate device memory
    hipMalloc((void **)&d_a, (i) * sizeof(double));
    hipMalloc((void **)&d_b, (i) * sizeof(double));

    hipMemcpy(d_a, h_a, (i) * sizeof(double), hipMemcpyHostToDevice);
    int grid_size = (i + BLOCK_SIZE - 1) / BLOCK_SIZE;
    // Warmup
    for (int j = 0; j < WARMUP; j++)
      cuda_vec_sclr_mul(d_a, d_b, 1 / 10, i, grid_size, BLOCK_SIZE);

    hipDeviceSynchronize();

    clock_t t = clock();
    for (int j = 0; j < TRAILS; j++)
      cuda_vec_sclr_mul(d_a, d_b, 1 / 10, i, grid_size, BLOCK_SIZE);
    hipDeviceSynchronize();
    t = clock() - t;

    hipFree(d_a), hipFree(d_b);
    fprintf(fp, "%s,%s,%u,%u,%e\n", "vec-sclr-mul", "cuda", 32, i,
            (double)t / (CLOCKS_PER_SEC * TRAILS));
    tfree(&h_a);
  }
  fclose(fp);
}

void vec_sclr_div_bench() {
  FILE *fp = open_file("vec-sclr-div-cuda");
  for (int i = 1e4; i < 1e7; i = inc(i)) {
    double *h_a = create_host_vec(i);

    double *d_a, *d_b;

    // Allocate device memory
    hipMalloc((void **)&d_a, (i) * sizeof(double));
    hipMalloc((void **)&d_b, (i) * sizeof(double));

    hipMemcpy(d_a, h_a, (i) * sizeof(double), hipMemcpyHostToDevice);
    int grid_size = (i + BLOCK_SIZE - 1) / BLOCK_SIZE;
    // Warmup
    for (int j = 0; j < WARMUP; j++)
      cuda_vec_sclr_div(d_a, d_b, 10, i, grid_size, BLOCK_SIZE);

    hipDeviceSynchronize();

    clock_t t = clock();
    for (int j = 0; j < TRAILS; j++)
      cuda_vec_sclr_div(d_a, d_b,  10, i, grid_size, BLOCK_SIZE);
    hipDeviceSynchronize();
    t = clock() - t;

    hipFree(d_a), hipFree(d_b);
    fprintf(fp, "%s,%s,%u,%u,%e\n", "vec-sclr-div", "cuda", 32, i,
            (double)t / (CLOCKS_PER_SEC * TRAILS));
    tfree(&h_a);
  }
  fclose(fp);
}

void calc_w_bench() {
  FILE *fp = open_file("calc_w_cuda");
  for (int i = 1e2; i < 3.7e4; i = inc(i)) {
    double *h_a = create_host_vec(i);
    double *h_b = create_host_vec(i * i);

    double *d_a, *d_b;

    // Allocate device memory
    hipMalloc((void **)&d_a, (i) * sizeof(double));
    hipMalloc((void **)&d_b, (i * i) * sizeof(double));

    hipMemcpy(d_a, h_a, (i) * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, (i * i) * sizeof(double), hipMemcpyHostToDevice);
    int grid_size = (i + BLOCK_SIZE - 1) / BLOCK_SIZE;
    // Warmup
    for (int j = 0; j < WARMUP; j++)
      cuda_calc_w(d_a, 10, d_b, 20, i - 1, i, grid_size, BLOCK_SIZE);

    hipDeviceSynchronize();

    clock_t t = clock();
    for (int j = 0; j < TRAILS; j++)
      cuda_calc_w(d_a, 10, d_b, 20, i - 1, i, grid_size, BLOCK_SIZE);
    hipDeviceSynchronize();
    t = clock() - t;

    hipFree(d_a), hipFree(d_b);
    fprintf(fp, "%s,%s,%u,%u,%e\n", "calc_w", "cuda", 32, i,
            (double)t / (CLOCKS_PER_SEC * TRAILS));
    tfree(&h_a);
    tfree(&h_b);
  }
  fclose(fp);
}

void vec_norm_bench() {
  FILE *fp = open_file("vec-norm");
  for (int i = 1e4; i < 1e7; i = inc(i)) {
    double *h_a = create_host_vec(i);
    double *d_a;
    hipMalloc((void **)&d_a, i * sizeof(double));
    hipMemcpy(d_a, h_a, i * sizeof(double), hipMemcpyHostToDevice);

    int grid_size = (i + BLOCK_SIZE - 1) / BLOCK_SIZE;

    // Warmup runs
    for (int j = 0; j < WARMUP; j++)
      cuda_vec_norm(d_a, i, grid_size, BLOCK_SIZE);

    // Measure time
    clock_t t = clock();
    for (int j = 0; j < TRAILS; j++)
      cuda_vec_norm(d_a, i, grid_size, BLOCK_SIZE);
    t = clock() - t;

    fprintf(fp, "%s,%s,%u,%u,%e\n", "vec-norm", "cuda", 32, i,
            (double)t / (CLOCKS_PER_SEC * TRAILS));

    hipFree(d_a);
    tfree(&h_a);
  }
  fclose(fp);
}

void vec_dot_bench() {
  FILE *fp = open_file("vec-dot");
  for (int i = 1e4; i < 1e7; i = inc(i)) {
    double *h_a = create_host_vec(i);
    double *h_b = create_host_vec(i);
    double *d_a ,*d_b;

    hipMalloc((void **)&d_a, i * sizeof(double));
    hipMemcpy(d_a, h_a, i * sizeof(double), hipMemcpyHostToDevice);

    hipMalloc((void **)&d_b, i * sizeof(double));
    hipMemcpy(d_b, h_b, i * sizeof(double), hipMemcpyHostToDevice);

    int grid_size = (i + BLOCK_SIZE - 1) / BLOCK_SIZE;

    // Warmup runs
    for (int j = 0; j < WARMUP; j++)
      cuda_vec_dot(d_a,d_b, i, grid_size, BLOCK_SIZE);

    // Measure time
    clock_t t = clock();
    for (int j = 0; j < TRAILS; j++)
      cuda_vec_dot(d_a,d_b, i, grid_size, BLOCK_SIZE);
    t = clock() - t;

    fprintf(fp, "%s,%s,%u,%u,%e\n", "vec-dot", "cuda", 32, i,
            (double)t / (CLOCKS_PER_SEC * TRAILS));

    hipFree(d_a);
    hipFree(d_b);
    tfree(&h_a);
    tfree(&h_b);
  }
  fclose(fp);
}

void mtx_col_copy_bench() {
  FILE *fp = open_file("mtx-col-copy");
  for (int i = 1e2; i < 3.7e4; i = inc(i)) {
    double *h_a = create_host_vec(i);
    double *d_a, *d_b;
    hipMalloc((void **)&d_a, i * sizeof(double));
    hipMalloc((void **)&d_b, i * i * sizeof(double));
    hipMemcpy(d_a, h_a, i * sizeof(double), hipMemcpyHostToDevice);

    int grid_size = (i + BLOCK_SIZE - 1) / BLOCK_SIZE;

    // Warmup runs
    for (int j = 0; j < WARMUP; j++)
      cuda_mtx_col_copy(d_a, d_b, 0, i, grid_size, BLOCK_SIZE);
    hipDeviceSynchronize();
    // Measure time
    clock_t t = clock();
    for (int j = 0; j < TRAILS; j++)
      cuda_mtx_col_copy(d_a, d_b, 0, i, grid_size, BLOCK_SIZE);
    hipDeviceSynchronize();
    t = clock() - t;

    fprintf(fp, "%s,%s,%u,%u,%e\n", "mtx-col-copy", "cuda", 32, i,
            (double)t / (CLOCKS_PER_SEC * TRAILS));

    hipFree(d_a), hipFree(d_b);
    tfree(&h_a);
  }
  fclose(fp);
}

void create_roofline() {
  FILE *fp = open_file("roofline_data");
  for (int i = 1e4; i < 1e7; i = inc(i)) {
    double *h_a = create_host_vec(i);

    double *d_a, *d_b;

    // Allocate device memory
    hipMalloc((void **)&d_a, (i) * sizeof(double));
    hipMalloc((void **)&d_b, (i) * sizeof(double));

    hipMemcpy(d_a, h_a, (i) * sizeof(double), hipMemcpyHostToDevice);

    int grid_size = (i + BLOCK_SIZE - 1) / BLOCK_SIZE;

    // Warmup d2d
    for (int j = 0; j < WARMUP; j++)
      cuda_d2d_mem_cpy(d_a, d_b, i, grid_size, BLOCK_SIZE);
    hipDeviceSynchronize();

    clock_t t = clock();
    for (int j = 0; j < TRAILS; j++)
      cuda_d2d_mem_cpy(d_a, d_b, i, grid_size, BLOCK_SIZE);
    hipDeviceSynchronize();
    t = clock() - t;

    hipFree(d_a), hipFree(d_b);
    fprintf(fp, "%s,%s,%u,%u,%e\n", "roofline", "cuda", 32, i,
            (double)t / (CLOCKS_PER_SEC * TRAILS));
    tfree(&h_a);
  }
  fclose(fp);
}

void spmv_bench() {
  FILE *fp = open_file("spmv_data");
  for (int i = 1e2; i < 1e4; i = inc(i)) {
    double *lap ,*vals, *h_orth_vec;
    int *row_ptrs, *columns, val_count;
    lap = (double *)calloc(i * i, sizeof(double));    
    create_lap(lap,i);
    lap_to_csr(lap, i, i, &row_ptrs, &columns, &vals, &val_count);
    h_orth_vec=create_host_vec(i);
    double *d_vals, *d_orth_mtx, *d_orth_vec, *d_w_vec;
    int *d_row_ptrs, *d_columns;

    // Allocate device memory
  hipMalloc((void **)&d_row_ptrs, (i + 1) * sizeof(int));
  hipMalloc((void **)&d_columns, (val_count) * sizeof(int));
  hipMalloc((void **)&d_vals, (val_count) * sizeof(double));
  hipMalloc((void **)&d_orth_vec, (i) * sizeof(double));
  hipMalloc((void **)&d_w_vec, (i) * sizeof(double));

  // H2D memory copy
  hipMemcpy(d_row_ptrs, row_ptrs, (i + 1) * sizeof(int),
             hipMemcpyHostToDevice);
  hipMemcpy(d_columns, columns, (val_count) * sizeof(int),
             hipMemcpyHostToDevice);
  hipMemcpy(d_vals, vals, (val_count) * sizeof(double),
             hipMemcpyHostToDevice);
  hipMemcpy(d_orth_vec, h_orth_vec, (i) * sizeof(double),
             hipMemcpyHostToDevice);

    int grid_size = (i + BLOCK_SIZE - 1) / BLOCK_SIZE;

    // Warmup d2d
    for (int j = 0; j < WARMUP; j++)
      cuda_spmv(d_row_ptrs, d_columns, d_vals, d_orth_vec, d_w_vec, i, i);
    hipDeviceSynchronize();

    clock_t t = clock();
    for (int j = 0; j < TRAILS; j++)
      cuda_spmv(d_row_ptrs, d_columns, d_vals, d_orth_vec, d_w_vec, i, i);
    hipDeviceSynchronize();
    t = clock() - t;

    // Free device memory
    hipFree(d_row_ptrs), hipFree(d_columns), hipFree(d_vals),
      hipFree(d_orth_vec), hipFree(d_w_vec);

    fprintf(fp, "%s,%s,%u,%u,%e\n", "spmv", "cuda", 32, i,
            (double)t / (CLOCKS_PER_SEC * TRAILS));


    tfree(&lap);
    tfree(&vals);
    tfree(&row_ptrs);
    tfree(&columns);
    tfree(&h_orth_vec);
    
  }
  fclose(fp);
}


void lanczos_bench(int argc, char *argv[]) {
  // vec_norm_bench();
  // calc_w_bench();
  // vec_sclr_div_bench();
  // vec_dot_bench();
  spmv_bench();
  // vec_sclr_mul_bench();
  // calc_w_bench();
  // vec_norm_bench()
}
