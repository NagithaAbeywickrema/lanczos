#include "hip/hip_runtime.h"
#include "bench.h"

void vec_sclr_mul_bench() {
  FILE *fp = open_file("vec-sclr-mul-cuda");
  for (int i = 1e4; i < 1e7; i = inc(i)) {
    double *h_a = create_host_vec(i);

    double *d_a, *d_b;

    // Allocate device memory
    hipMalloc((void **)&d_a, (i) * sizeof(double));
    hipMalloc((void **)&d_b, (i) * sizeof(double));

    hipMemcpy(d_a, h_a, (i) * sizeof(double), hipMemcpyHostToDevice);
    int grid_size = (i + BLOCK_SIZE - 1) / BLOCK_SIZE;
    // Warmup
    for (int j = 0; j < WARMUP; j++)
      cuda_vec_sclr_mul(d_a, d_b, 1 / 10, i, grid_size, BLOCK_SIZE);

    hipDeviceSynchronize();

    clock_t t = clock();
    for (int j = 0; j < TRAILS; j++)
      cuda_vec_sclr_mul(d_a, d_b, 1 / 10, i, grid_size, BLOCK_SIZE);
    hipDeviceSynchronize();
    t = clock() - t;

    double *w_vec = (double *)calloc(i, sizeof(double));
    double *out = (double *)calloc(i, sizeof(double));
    serial_vec_sclr_mul(h_a, w_vec, 1 / 10, i);

    hipMemcpy(out, d_b, i * sizeof(double), hipMemcpyDeviceToHost);
    for (int k = 0; k < i; k++) {
      assert(fabs(w_vec[k] - out[k]) < EPS);
    }

    hipFree(d_a), hipFree(d_b);
    fprintf(fp, "%s,%s,%u,%u,%e\n", "vec-sclr-mul", "cuda", 32, i,
            (double)t / (CLOCKS_PER_SEC * TRAILS));
    tfree(&h_a);
  }
  fclose(fp);
}

void vec_sclr_div_bench() {
  FILE *fp = open_file("vec-sclr-div-cuda");
  for (int i = 1e4; i < 1e7; i = inc(i)) {
    double *h_a = create_host_vec(i);

    double *d_a, *d_b;

    // Allocate device memory
    hipMalloc((void **)&d_a, (i) * sizeof(double));
    hipMalloc((void **)&d_b, (i) * sizeof(double));

    hipMemcpy(d_a, h_a, (i) * sizeof(double), hipMemcpyHostToDevice);
    int grid_size = (i + BLOCK_SIZE - 1) / BLOCK_SIZE;
    // Warmup
    for (int j = 0; j < WARMUP; j++)
      cuda_vec_sclr_div(d_a, d_b, 10, i, grid_size, BLOCK_SIZE);

    hipDeviceSynchronize();

    clock_t t = clock();
    for (int j = 0; j < TRAILS; j++)
      cuda_vec_sclr_div(d_a, d_b, 10, i, grid_size, BLOCK_SIZE);
    hipDeviceSynchronize();
    t = clock() - t;

    double *w_vec = (double *)calloc(i, sizeof(double));
    double *out = (double *)calloc(i, sizeof(double));
    serial_vec_sclr_div(h_a, w_vec, 10, i);

    hipMemcpy(out, d_b, i * sizeof(double), hipMemcpyDeviceToHost);
    for (int k = 0; k < i; k++) {
      assert(fabs(w_vec[k] - out[k]) < EPS);
    }

    hipFree(d_a), hipFree(d_b);
    fprintf(fp, "%s,%s,%u,%u,%e\n", "vec-sclr-div", "cuda", 32, i,
            (double)t / (CLOCKS_PER_SEC * TRAILS));
    tfree(&h_a);
  }
  fclose(fp);
}

void calc_w_bench() {
  FILE *fp = open_file("calc_w_cuda");
  for (int i = 1e2; i < 3.7e4; i = inc(i)) {
    double *h_a = create_host_vec(i);
    double *h_b = create_host_vec(i * i);

    double *d_a, *d_b;

    // Allocate device memory
    hipMalloc((void **)&d_a, (i) * sizeof(double));
    hipMalloc((void **)&d_b, (i * i) * sizeof(double));

    hipMemcpy(d_a, h_a, (i) * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, (i * i) * sizeof(double), hipMemcpyHostToDevice);
    int grid_size = (i + BLOCK_SIZE - 1) / BLOCK_SIZE;
    // Warmup
    for (int j = 0; j < WARMUP; j++)
      cuda_calc_w(d_a, 10, d_b, 20, i - 1, i, grid_size, BLOCK_SIZE);

    hipDeviceSynchronize();

    clock_t t = clock();
    for (int j = 0; j < TRAILS; j++)
      cuda_calc_w(d_a, 10, d_b, 20, i - 1, i, grid_size, BLOCK_SIZE);
    hipDeviceSynchronize();
    t = clock() - t;

    double *out = (double *)calloc(i, sizeof(double));
    serial_calc_w(h_a, 10, h_b, 20, i - 1, i);

    hipMemcpy(out, d_b, i * sizeof(double), hipMemcpyDeviceToHost);
    for (int k = 0; k < i; k++) {
      assert(fabs(h_b[k] - out[k]) < EPS);
    }

    hipFree(d_a), hipFree(d_b);
    fprintf(fp, "%s,%s,%u,%u,%e\n", "calc_w", "cuda", 32, i,
            (double)t / (CLOCKS_PER_SEC * TRAILS));
    tfree(&h_a);
    tfree(&h_b);
  }
  fclose(fp);
}

void vec_norm_bench() {
  FILE *fp = open_file("vec-norm");
  for (int i = 1e4; i < 1e7; i = inc(i)) {
    double *h_a = create_host_vec(i);
    double *d_a;

    double out;
    hipMalloc((void **)&d_a, i * sizeof(double));
    hipMemcpy(d_a, h_a, i * sizeof(double), hipMemcpyHostToDevice);

    int grid_size = (i + BLOCK_SIZE - 1) / BLOCK_SIZE;

    // Warmup runs
    for (int j = 0; j < WARMUP; j++)
      cuda_vec_norm(d_a, i, grid_size, BLOCK_SIZE);

    // Measure time
    clock_t t = clock();
    for (int j = 0; j < TRAILS; j++)
      out = cuda_vec_norm(d_a, i, grid_size, BLOCK_SIZE);
    t = clock() - t;

    double norm_val = serial_vec_norm(h_a, i);

    assert(fabs(norm_val - out) < EPS);

    fprintf(fp, "%s,%s,%u,%u,%e\n", "vec-norm", "cuda", 32, i,
            (double)t / (CLOCKS_PER_SEC * TRAILS));

    hipFree(d_a);
    tfree(&h_a);
  }
  fclose(fp);
}

void vec_dot_bench() {
  FILE *fp = open_file("vec-dot");
  for (int i = 1e4; i < 1e7; i = inc(i)) {
    double *h_a = create_host_vec(i);
    double *h_b = create_host_vec(i);
    double *d_a, *d_b;
    double out;
    hipMalloc((void **)&d_a, i * sizeof(double));
    hipMemcpy(d_a, h_a, i * sizeof(double), hipMemcpyHostToDevice);

    hipMalloc((void **)&d_b, i * sizeof(double));
    hipMemcpy(d_b, h_b, i * sizeof(double), hipMemcpyHostToDevice);

    int grid_size = (i + BLOCK_SIZE - 1) / BLOCK_SIZE;

    // Warmup runs
    for (int j = 0; j < WARMUP; j++)
      cuda_vec_dot(d_a, d_b, i, grid_size, BLOCK_SIZE);

    // Measure time
    clock_t t = clock();
    for (int j = 0; j < TRAILS; j++)
      out = cuda_vec_dot(d_a, d_b, i, grid_size, BLOCK_SIZE);
    t = clock() - t;

    double dot = serial_vec_dot(h_a, h_b, i);

    assert(fabs(dot - out) < EPS);

    fprintf(fp, "%s,%s,%u,%u,%e\n", "vec-dot", "cuda", 32, i,
            (double)t / (CLOCKS_PER_SEC * TRAILS));

    hipFree(d_a);
    hipFree(d_b);
    tfree(&h_a);
    tfree(&h_b);
  }
  fclose(fp);
}

void mtx_col_copy_bench() {
  FILE *fp = open_file("mtx-col-copy");
  for (int i = 1e2; i < 3.7e4; i = inc(i)) {
    double *h_a = create_host_vec(i);
    double *d_a, *d_b;
    hipMalloc((void **)&d_a, i * sizeof(double));
    hipMalloc((void **)&d_b, i * i * sizeof(double));
    hipMemcpy(d_a, h_a, i * sizeof(double), hipMemcpyHostToDevice);

    int grid_size = (i + BLOCK_SIZE - 1) / BLOCK_SIZE;

    // Warmup runs
    for (int j = 0; j < WARMUP; j++)
      cuda_mtx_col_copy(d_a, d_b, 0, i, grid_size, BLOCK_SIZE);
    hipDeviceSynchronize();
    // Measure time
    clock_t t = clock();
    for (int j = 0; j < TRAILS; j++)
      cuda_mtx_col_copy(d_a, d_b, 0, i, grid_size, BLOCK_SIZE);
    hipDeviceSynchronize();
    t = clock() - t;

    double *mtx = (double *)calloc(i * i, sizeof(double));
    double *out = (double *)calloc(i, sizeof(double));
    serial_mtx_col_copy(h_a, mtx, 0, i);

    hipMemcpy(out, d_b, i * i * sizeof(double), hipMemcpyDeviceToHost);
    for (int k = 0; k < i * i; k++) {
      assert(fabs(mtx[k] - out[k]) < EPS);
    }

    fprintf(fp, "%s,%s,%u,%u,%e\n", "mtx-col-copy", "cuda", 32, i,
            (double)t / (CLOCKS_PER_SEC * TRAILS));

    hipFree(d_a), hipFree(d_b);
    tfree(&h_a);
  }
  fclose(fp);
}

void create_roofline() {
  FILE *fp = open_file("roofline_data");
  for (int i = 1e4; i < 1e7; i = inc(i)) {
    double *h_a = create_host_vec(i);

    double *d_a, *d_b;

    // Allocate device memory
    hipMalloc((void **)&d_a, (i) * sizeof(double));
    hipMalloc((void **)&d_b, (i) * sizeof(double));

    hipMemcpy(d_a, h_a, (i) * sizeof(double), hipMemcpyHostToDevice);

    int grid_size = (i + BLOCK_SIZE - 1) / BLOCK_SIZE;

    // Warmup d2d
    for (int j = 0; j < WARMUP; j++)
      cuda_d2d_mem_cpy(d_a, d_b, i, grid_size, BLOCK_SIZE);
    hipDeviceSynchronize();

    clock_t t = clock();
    for (int j = 0; j < TRAILS; j++)
      cuda_d2d_mem_cpy(d_a, d_b, i, grid_size, BLOCK_SIZE);
    hipDeviceSynchronize();
    t = clock() - t;

    double *out = (double *)calloc(i, sizeof(double));

    hipMemcpy(out, d_b, i * sizeof(double), hipMemcpyDeviceToHost);
    for (int k = 0; k < i; k++) {
      assert(fabs(h_a[k] - out[k]) < EPS);
    }

    hipFree(d_a), hipFree(d_b);
    fprintf(fp, "%s,%s,%u,%u,%e\n", "roofline", "cuda", 32, i,
            (double)t / (CLOCKS_PER_SEC * TRAILS));
    tfree(&h_a);
  }
  fclose(fp);
}

void spmv_bench() {
  FILE *fp = open_file("spmv_data");
  for (int i = 1e2; i < 1e4; i = inc(i)) {
    double *lap, *vals, *h_orth_vec;
    int *row_ptrs, *columns, val_count;
    lap = (double *)calloc(i * i, sizeof(double));
    create_lap(lap, i, 100);
    lap_to_csr(lap, i, i, &row_ptrs, &columns, &vals, &val_count);
    h_orth_vec = create_host_vec(i);
    double *d_vals, *d_orth_mtx, *d_orth_vec, *d_w_vec;
    int *d_row_ptrs, *d_columns;

    // Allocate device memory
    hipMalloc((void **)&d_row_ptrs, (i + 1) * sizeof(int));
    hipMalloc((void **)&d_columns, (val_count) * sizeof(int));
    hipMalloc((void **)&d_vals, (val_count) * sizeof(double));
    hipMalloc((void **)&d_orth_vec, (i) * sizeof(double));
    hipMalloc((void **)&d_w_vec, (i) * sizeof(double));

    // H2D memory copy
    hipMemcpy(d_row_ptrs, row_ptrs, (i + 1) * sizeof(int),
               hipMemcpyHostToDevice);
    hipMemcpy(d_columns, columns, (val_count) * sizeof(int),
               hipMemcpyHostToDevice);
    hipMemcpy(d_vals, vals, (val_count) * sizeof(double),
               hipMemcpyHostToDevice);
    hipMemcpy(d_orth_vec, h_orth_vec, (i) * sizeof(double),
               hipMemcpyHostToDevice);

    int grid_size = (i + BLOCK_SIZE - 1) / BLOCK_SIZE;

    // Warmup d2d
    for (int j = 0; j < WARMUP; j++)
      cuda_spmv(d_row_ptrs, d_columns, d_vals, d_orth_vec, d_w_vec, i, i,
                grid_size, BLOCK_SIZE);
    hipDeviceSynchronize();

    clock_t t = clock();
    for (int j = 0; j < TRAILS; j++)
      cuda_spmv(d_row_ptrs, d_columns, d_vals, d_orth_vec, d_w_vec, i, i,
                grid_size, BLOCK_SIZE);
    hipDeviceSynchronize();
    t = clock() - t;

    double *w_vec = (double *)calloc(i, sizeof(double));
    double *out = (double *)calloc(i, sizeof(double));
    serial_spmv(row_ptrs, columns, vals, h_orth_vec, w_vec, i, i);

    hipMemcpy(out, d_w_vec, i * sizeof(double), hipMemcpyDeviceToHost);
    for (int k = 0; k < i; k++) {
      assert(fabs(w_vec[k] - out[k]) < EPS);
    }
    // Free device memory
    hipFree(d_row_ptrs), hipFree(d_columns), hipFree(d_vals),
        hipFree(d_orth_vec), hipFree(d_w_vec);

    fprintf(fp, "%s,%s,%u,%u,%e,%u\n", "spmv", "cuda", 32, i,
            (double)t / (CLOCKS_PER_SEC * TRAILS), val_count);

    tfree(&lap);
    tfree(&vals);
    tfree(&row_ptrs);
    tfree(&columns);
    tfree(&h_orth_vec);
  }
  fclose(fp);
}

void lanczos_bench(int argc, char *argv[]) {
  vec_sclr_mul_bench();
  vec_sclr_div_bench();
  calc_w_bench();
  vec_norm_bench();
  vec_dot_bench();
  mtx_col_copy_bench();
  create_roofline();
  spmv_bench();
}
