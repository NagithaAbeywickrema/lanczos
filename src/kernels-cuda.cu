#include "hip/hip_runtime.h"
#include "kernels.h"
#include <math.h>

#define BLOCK_SIZE 32

__global__ void cuda_vec_dot_knl(double *a_vec, double *b_vec, double *result,
                                 const unsigned size) {
  extern __shared__ double shared_data[];
  const unsigned tid = blockIdx.x * blockDim.x + threadIdx.x;

  if (tid < size)
    shared_data[threadIdx.x] = a_vec[tid] * b_vec[tid];
  else
    shared_data[threadIdx.x] = 0.0;

  __syncthreads();

  for (unsigned stride = blockDim.x >> 1; stride > 0; stride >>= 1) {
    if (threadIdx.x < stride)
      shared_data[threadIdx.x] += shared_data[threadIdx.x + stride];

    __syncthreads();
  }

  if (threadIdx.x == 0)
    result[blockIdx.x] = shared_data[0];
}

__global__ void cuda_vec_sclr_div_knl(double *a_vec, double *out_vec,
                                      const double sclr, const unsigned size) {
  const unsigned tid = blockIdx.x * blockDim.x + threadIdx.x;

  if (tid < size)
    out_vec[tid] = a_vec[tid] / sclr;
}

__global__ void cuda_vec_sclr_mul_knl(double *a_vec, double *out_vec,
                                      const double sclr, const unsigned size) {
  const unsigned tid = blockIdx.x * blockDim.x + threadIdx.x;

  if (tid < size)
    out_vec[tid] = a_vec[tid] * sclr;
}

__global__ void cuda_d2d_mem_cpy_knl(double *a, double *b, unsigned int size) {
  const unsigned tid = blockIdx.x * blockDim.x + threadIdx.x;

  if (tid < size)
    b[tid] = a[tid];
}

__global__ void cuda_mtx_col_copy_knl(double *vec, double *mtx,
                                      const unsigned col_index,
                                      const unsigned size) {
  const unsigned tid = blockIdx.x * blockDim.x + threadIdx.x;

  if (tid < size)
    mtx[tid + size * col_index] = vec[tid];
}

__global__ void cuda_mtx_vec_mul_knl(double *a_mtx, double *b_vec,
                                     double *out_vec, const unsigned num_rows,
                                     const unsigned num_cols) {
  const unsigned row = blockIdx.x * blockDim.x + threadIdx.x;

  if (row < num_rows) {
    double dot = 0;
    for (unsigned k = 0; k < num_cols; k++)
      dot += a_mtx[row * num_cols + k] * b_vec[k];
    out_vec[row] = dot;
  }
}

__global__ void cuda_spmv_knl(unsigned *a_row_ptrs, unsigned *a_columns,
                              double *a_vals, double *b_vec, double *out_vec,
                              const unsigned num_rows,
                              const unsigned num_cols) {
  const unsigned row = blockIdx.x * blockDim.x + threadIdx.x;

  if (row < num_rows) {
    unsigned start = a_row_ptrs[row];
    unsigned end = a_row_ptrs[row + 1];
    double dot = 0;
    // Add each element in the row
    for (unsigned j = start; j < end; j++)
      dot += a_vals[j] * b_vec[a_columns[j]];
    out_vec[row] = dot;
  }
}

__global__ void cuda_calc_w_init_knl(double *w_vec, const double alpha,
                                     double *orth_mtx, const unsigned col_index,
                                     const unsigned size) {
  const unsigned tid = blockIdx.x * blockDim.x + threadIdx.x;

  if (tid < size) {
    w_vec[tid] = w_vec[tid] - alpha * orth_mtx[tid + size * col_index];
  }
}

__global__ void cuda_calc_w_knl(double *w_vec, const double alpha,
                                double *orth_mtx, const double beta,
                                const unsigned col_index, const unsigned size) {
  const unsigned tid = blockIdx.x * blockDim.x + threadIdx.x;

  if (tid < size) {
    w_vec[tid] = w_vec[tid] - alpha * orth_mtx[tid + size * col_index] -
                 beta * orth_mtx[tid + size * (col_index - 1)];
  }
}

double cuda_vec_dot(double *d_a_vec, double *d_b_vec, const unsigned size) {
  const unsigned grid_size = (size + BLOCK_SIZE - 1) / BLOCK_SIZE;
  const unsigned shared_data_size = BLOCK_SIZE * sizeof(double);

  double *d_result;

  hipMalloc((void **)&d_result, grid_size * sizeof(double));

  cuda_vec_dot_knl<<<grid_size, BLOCK_SIZE, shared_data_size>>>(
      d_a_vec, d_b_vec, d_result, size);

  hipDeviceSynchronize();

  double *interim_results = (double *)calloc(grid_size, sizeof(double));
  hipMemcpy(interim_results, d_result, grid_size * sizeof(double),
             hipMemcpyDeviceToHost);

  double result = 0.0;
  for (unsigned i = 0; i < grid_size; i++) {
    result += interim_results[i];
  }

  hipFree(d_result), free(interim_results);

  return result;
}

double cuda_vec_norm(double *d_a_vec, const unsigned size) {
  double sum_of_prod = cuda_vec_dot(d_a_vec, d_a_vec, size);
  return sqrt(sum_of_prod);
}

void cuda_vec_sclr_div(double *d_a_vec, double *d_out_vec, const double sclr,
                       const unsigned size,const unsigned grid_size, const unsigned block_size) {

  cuda_vec_sclr_div_knl<<<grid_size, block_size>>>(d_a_vec, d_out_vec, sclr,
                                                   size);
}

void cuda_vec_sclr_mul(double *d_a_vec, double *d_out_vec, const double sclr,
                       const unsigned size,const unsigned grid_size, const unsigned block_size) {

  cuda_vec_sclr_mul_knl<<<grid_size, block_size>>>(d_a_vec, d_out_vec, sclr,
                                                   size);
}


void cuda_d2d_mem_cpy(double *a, double *b, unsigned int size,const unsigned grid_size, const unsigned block_size) {

  cuda_d2d_mem_cpy_knl<<<grid_size, block_size>>>(a, b, size);
}

void cuda_mtx_col_copy(double *d_vec, double *d_mtx, const unsigned col_index,
                       const unsigned size) {
  const unsigned grid_size = (size + BLOCK_SIZE - 1) / BLOCK_SIZE;

  cuda_mtx_col_copy_knl<<<grid_size, BLOCK_SIZE>>>(d_vec, d_mtx, col_index,
                                                   size);

  hipDeviceSynchronize();
}

void cuda_mtx_vec_mul(double *d_a_mtx, double *d_b_vec, double *d_out_vec,
                      const unsigned num_rows, const unsigned num_cols) {
  const unsigned grid_size = (num_rows + BLOCK_SIZE - 1) / BLOCK_SIZE;

  cuda_mtx_vec_mul_knl<<<grid_size, BLOCK_SIZE>>>(d_a_mtx, d_b_vec, d_out_vec,
                                                  num_rows, num_cols);

  hipDeviceSynchronize();
}

void cuda_spmv(unsigned *d_a_row_ptrs, unsigned *d_a_columns, double *d_a_vals,
               double *d_b_vec, double *d_out_vec, const unsigned num_rows,
               const unsigned num_cols) {
  const unsigned grid_size = (num_rows + BLOCK_SIZE - 1) / BLOCK_SIZE;

  cuda_spmv_knl<<<grid_size, BLOCK_SIZE>>>(d_a_row_ptrs, d_a_columns, d_a_vals,
                                           d_b_vec, d_out_vec, num_rows,
                                           num_cols);

  hipDeviceSynchronize();
}

void cuda_calc_w_init(double *d_w_vec, const double alpha, double *d_orth_mtx,
                      const unsigned col_index, const unsigned size) {
  const unsigned grid_size = (size + BLOCK_SIZE - 1) / BLOCK_SIZE;

  cuda_calc_w_init_knl<<<grid_size, BLOCK_SIZE>>>(d_w_vec, alpha, d_orth_mtx,
                                                  col_index, size);

  hipDeviceSynchronize();
}

void cuda_calc_w(double *d_w_vec, const double alpha, double *d_orth_mtx,
                 const double beta, const unsigned col_index,
                 const unsigned size,const unsigned grid_size, const unsigned block_size) {

  cuda_calc_w_knl<<<grid_size, block_size>>>(d_w_vec, alpha, d_orth_mtx, beta,
                                             col_index, size);

  hipDeviceSynchronize();
}
